#include "hip/hip_runtime.h"
#include "t.h"
#include <time.h>

#define DIM 7
#define REP 1000

__device__ int calc(int arg1, int arg2)
{
	return (arg1+arg2)/2;
}
__global__ void MatAddKernel(int* ma, int* mb, int* mc)
{
	unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
//	__syncthreads();
	mc[tid] = calc(ma[tid], mb[tid]);
//	__syncthreads;
}
extern "C" void MatAdd()
{
	unsigned int nBlock=5,nThread=3;
	unsigned int memSize = sizeof(int) * nBlock * nThread;
	int *ma, *mb, *mc;
	
	//host data
	ma = (int*)malloc(memSize);
	mb = (int*)malloc(memSize);
	mc = (int*)malloc(memSize);

	int *dma, *dmb, *dmc;
	
	//device data
	hipMalloc((void**)&dma, memSize);
	hipMalloc((void**)&dmb, memSize);
	hipMalloc((void**)&dmc, memSize);
	for(unsigned int i = 0; i < nBlock * nThread; i++)
		ma[i] =	mb[i] = i;

	hipMemcpy(dma, ma, memSize, hipMemcpyHostToDevice);
	hipMemcpy(dmb, mb, memSize, hipMemcpyHostToDevice);
	dim3 grid( nBlock, 1, 1);
	dim3 threads( nThread, 1, 1);

	MatAddKernel<<<grid, threads>>>(dma, dmb, dmc);
	hipMemcpy(mc, dmc, memSize, hipMemcpyDeviceToHost);
	for(int i=0; i<nBlock;i++)
	  {
		for(int j=0; j<nThread;j++)
	 	{
			int id = i * nThread + j;
			printf("ma:%d  mb:%d  mc%d   ",ma[id],mb[id],mc[id]);
	  	}
		printf("\n");
	 } 
	free(ma);
	free(mb);
	free(mc);
	hipFree(dma);
	hipFree(dmb);
	hipFree(dmc);
}
